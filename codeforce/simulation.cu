#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <SDL2/SDL.h>

const int N = 64;  // Grid size
const int WINDOW_SIZE = 512;

#define IX(i, j) ((i) + (N + 2) * (j))

__global__ void add_source(int size, float *x, float *s, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        x[i] += dt * s[i];
    }
}

__global__ void diffuse(int size, int b, float *x, float *x0, float diff, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float a = dt * diff * N * N;
        x[IX(i % N, i / N)] = (x0[IX(i % N, i / N)] + a * (x[IX((i % N) - 1, i / N)] + x[IX((i % N) + 1, i / N)] + x[IX(i % N, (i / N) - 1)] + x[IX(i % N, (i / N) + 1)])) / (1 + 4 * a);
    }
}

__global__ void advect(int size, int b, float *d, float *d0, float *u, float *v, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float dt0 = dt * N;
        float x = (i % N) - dt0 * u[IX((i % N), (i / N))];
        float y = (i / N) - dt0 * v[IX((i % N), (i / N))];
        if (x < 0.5f) x = 0.5f;
        if (x > N + 0.5f) x = N + 0.5f;
        int i0 = (int)x;
        int i1 = i0 + 1;
        if (y < 0.5f) y = 0.5f;
        if (y > N + 0.5f) y = N + 0.5f;
        int j0 = (int)y;
        int j1 = j0 + 1;
        float s1 = x - i0;
        float s0 = 1 - s1;
        float t1 = y - j0;
        float t0 = 1 - t1;
        d[IX(i % N, i / N)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) + s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    }
}

__global__ void project(int size, float *u, float *v, float *p, float *div) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        div[IX(i % N, i / N)] = -0.5f * (u[IX((i % N) + 1, i / N)] - u[IX((i % N) - 1, i / N)] + v[IX(i % N, (i / N) + 1)] - v[IX(i % N, (i / N) - 1)]) / N;
        p[IX(i % N, i / N)] = 0;
    }
}

void step(float *u, float *v, float *u_prev, float *v_prev, float *dens, float *dens_prev, float visc, float diff, float dt) {
    int size = (N + 2) * (N + 2);
    add_source<<<(size + 255) / 256, 256>>>(size, u, u_prev, dt);
    add_source<<<(size + 255) / 256, 256>>>(size, v, v_prev, dt);
    add_source<<<(size + 255) / 256, 256>>>(size, dens, dens_prev, dt);

    diffuse<<<(size + 255) / 256, 256>>>(size, 1, u, u_prev, visc, dt);
    diffuse<<<(size + 255) / 256, 256>>>(size, 2, v, v_prev, visc, dt);
    project<<<(size + 255) / 256, 256>>>(size, u, v, u_prev, v_prev);

    advect<<<(size + 255) / 256, 256>>>(size, 1, u, u_prev, u_prev, v_prev, dt);
    advect<<<(size + 255) / 256, 256>>>(size, 2, v, v_prev, u_prev, v_prev, dt);
    project<<<(size + 255) / 256, 256>>>(size, u, v, u_prev, v_prev);

    diffuse<<<(size + 255) / 256, 256>>>(size, 0, dens, dens_prev, diff, dt);
    advect<<<(size + 255) / 256, 256>>>(size, 0, dens, dens_prev, u, v, dt);
}

int main() {
    int size = (N + 2) * (N + 2) * sizeof(float);

    float *u, *v, *u_prev, *v_prev, *dens, *dens_prev;
    hipMallocManaged(&u, size);
    hipMallocManaged(&v, size);
    hipMallocManaged(&u_prev, size);
    hipMallocManaged(&v_prev, size);
    hipMallocManaged(&dens, size);
    hipMallocManaged(&dens_prev, size);

    memset(u, 0, size);
    memset(v, 0, size);
    memset(u_prev, 0, size);
    memset(v_prev, 0, size);
    memset(dens, 0, size);
    memset(dens_prev, 0, size);

    // Simulation parameters
    float dt = 0.1f;
    float diff = 0.0f;
    float visc = 0.0f;

    // Add some initial density
    for (int i = 20; i <= 40; i++) {
        for (int j = 20; j <= 40; j++) {
            dens[IX(i, j)] = 100.0f;
        }
    }

    // Initialize SDL
    SDL_Init(SDL_INIT_VIDEO);
    SDL_Window *window = SDL_CreateWindow("Fluid Simulation",
                                          SDL_WINDOWPOS_CENTERED,
                                          SDL_WINDOWPOS_CENTERED,
                                          WINDOW_SIZE,
                                          WINDOW_SIZE,
                                          SDL_WINDOW_SHOWN);
    SDL_Renderer *renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);

    SDL_Texture *texture = SDL_CreateTexture(renderer,
                                             SDL_PIXELFORMAT_RGBA8888,
                                             SDL_TEXTUREACCESS_STREAMING,
                                             WINDOW_SIZE,
                                             WINDOW_SIZE);

    for (int frame = 0; frame < 100; frame++) {
        step(u, v, u_prev, v_prev, dens, dens_prev, visc, diff, dt);

        // Update SDL texture
        uint32_t *pixels = new uint32_t[WINDOW_SIZE * WINDOW_SIZE];
        for (int y = 0; y < N; y++) {
            for (int x = 0; x < N; x++) {
                float density = dens[IX(x, y)];
                uint8_t color = (density > 0) ? (uint8_t)fminf(255, density) : 0;
                pixels[y * N + x] = SDL_MapRGBA(SDL_PIXELFORMAT_RGBA8888, color, 0, 0, 255);
            }
        }
        SDL_UpdateTexture(texture, NULL, pixels, WINDOW_SIZE * sizeof(uint32_t));
        delete[] pixels;

        SDL_RenderClear(renderer);
        SDL_RenderCopy(renderer, texture, NULL, NULL);
        SDL_RenderPresent(renderer);

        // Handle events
        SDL_Event event;
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                goto end;
            }
        }
        SDL_Delay(100); // Delay to control frame rate
    }

end:
    // Cleanup
    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    hipFree(u);
    hipFree(v);
    hipFree(u_prev);
    hipFree(v_prev);
    hipFree(dens);
    hipFree(dens_prev);

    return 0;
}
